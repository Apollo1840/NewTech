
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  int index = threadIdx.x;
  int stride = blockDim.x; // how big is one thread
  for (int i = index; i < n; i += stride)
      y[i] = x[i] + y[i];
}

int main(void)
{
  int N = 1<<20; // 1M elements
  float *x, *y;

  // Allocate Unified Memory 
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // 256 Threads
  add<<<1, 256>>>(N, x, y);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Free memory
  hipFree(x);
  hipFree(y);
  
  return 0;
}